#include "hip/hip_runtime.h"
// Homework 2
// Image Blurring
//
// In this homework we are blurring an image. To do this, imagine that we have
// a square array of weight values. For each pixel in the image, imagine that we
// overlay this square array of weights on top of the image such that the center
// of the weight array is aligned with the current pixel. To compute a blurred
// pixel value, we multiply each pair of numbers that line up. In other words, we
// multiply each weight with the pixel underneath it. Finally, we add up all of the
// multiplied numbers and assign that value to our output for the current pixel.
// We repeat this process for all the pixels in the image.

// To help get you started, we have included some useful notes here.

//****************************************************************************

// For a color image that has multiple channels, we suggest separating
// the different color channels so that each color is stored contiguously
// instead of being interleaved. This will simplify your code.

// That is instead of RGBARGBARGBARGBA... we suggest transforming to three
// arrays (as in the previous homework we ignore the alpha channel again):
//  1) RRRRRRRR...
//  2) GGGGGGGG...
//  3) BBBBBBBB...
//
// The original layout is known an Array of Structures (AoS) whereas the
// format we are converting to is known as a Structure of Arrays (SoA).

// As a warm-up, we will ask you to write the kernel that performs this
// separation. You should then write the "meat" of the assignment,
// which is the kernel that performs the actual blur. We provide code that
// re-combines your blurred results for each color channel.

//****************************************************************************

// You must fill in the gaussian_blur kernel to perform the blurring of the
// inputChannel, using the array of weights, and put the result in the outputChannel.

// Here is an example of computing a blur, using a weighted average, for a single
// pixel in a small image.
//
// Array of weights:
//
//  0.0  0.2  0.0
//  0.2  0.2  0.2
//  0.0  0.2  0.0
//
// Image (note that we align the array of weights to the center of the box):
//
//    1  2  5  2  0  3
//       -------
//    3 |2  5  1| 6  0       0.0*2 + 0.2*5 + 0.0*1 +
//      |       |
//    4 |3  6  2| 1  4   ->  0.2*3 + 0.2*6 + 0.2*2 +   ->  3.2
//      |       |
//    0 |4  0  3| 4  2       0.0*4 + 0.2*0 + 0.0*3
//       -------
//    9  6  5  0  3  9
//
//         (1)                         (2)                 (3)
//
// A good starting place is to map each thread to a pixel as you have before.
// Then every thread can perform steps 2 and 3 in the diagram above
// completely independently of one another.

// Note that the array of weights is square, so its height is the same as its width.
// We refer to the array of weights as a filter, and we refer to its width with the
// variable filterWidth.

//****************************************************************************

// Your homework submission will be evaluated based on correctness and speed.
// We test each pixel against a reference solution. If any pixel differs by
// more than some small threshold value, the system will tell you that your
// solution is incorrect, and it will let you try again.

// Once you have gotten that working correctly, then you can think about using
// shared memory and having the threads cooperate to achieve better performance.

//****************************************************************************

// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//
// Writing code the safe way requires slightly more typing, but is very helpful for
// catching mistakes. If you write code the unsafe way and you make a mistake, then
// any subsequent kernels won't compute anything, and it will be hard to figure out
// why. Writing code the safe way will inform you as soon as you make a mistake.

// Finally, remember to free the memory you allocate at the end of the function.

//****************************************************************************

#include "utils.h"
#include <stdio.h>
#include <algorithm>
#include <cassert>
#include <hip/hip_runtime.h>


#define gpuErrchk() { gpuAssert(__FILE__, __LINE__); }
inline void gpuAssert(const char* file, int line, bool abort = true)
{
    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}





__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO
  
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
  int outputRow = threadIdx.y + blockIdx.y * blockDim.y;
  int outputCol = threadIdx.x + blockIdx.x * blockDim.x;

  if (outputRow >= numRows || outputCol >=numCols){
    return;
  }

  unsigned int image1DIndexOutput = numCols * outputRow + outputCol;
  
  float outputVal = 0.0f;

  for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
    for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
      
      int image_r = outputRow + filter_r;
      image_r = image_r > 0 ? image_r :0;
      image_r = image_r < numRows ? image_r : numRows -1;     

      int image_c = outputCol + filter_c;
      image_c = image_c > 0 ? image_c :0;
      image_c = image_c < numCols ? image_c : numCols -1;

      int image1DIdx = numCols * image_r + image_c;
      int filter1DIdx = (filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2;

      outputVal += inputChannel[image1DIdx] * filter[filter1DIdx];
    }
  }

  outputChannel[image1DIndexOutput] = outputVal;

}

__global__
void gaussian_blur_sh_filter(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO
  
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
  int outputRow = threadIdx.y + blockIdx.y * blockDim.y;
  int outputCol = threadIdx.x + blockIdx.x * blockDim.x;

  if (outputRow >= numRows || outputCol >=numCols){
    return;
  }

  extern __shared__ float sh_filter[];

  int filter_r = threadIdx.y;
  int filter_c = threadIdx.x;

  if (filter_r < filterWidth && filter_c < filterWidth){
    //for (int filter_r = 0; filter_r < filterWidth; ++filter_r) {
    //  for (int filter_c = 0; filter_c < filterWidth; ++filter_c) {
        int filterIdx = filter_r * filterWidth + filter_c;
        sh_filter[filterIdx] = filter[filterIdx];
    //  }
    //}
  }

  __syncthreads();

  unsigned int image1DIndexOutput = numCols * outputRow + outputCol;
  
  float outputVal = 0.0f;

  for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
    for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
      
      int image_r = outputRow + filter_r;
      image_r = image_r > 0 ? image_r :0;
      image_r = image_r < numRows ? image_r : numRows -1;     

      int image_c = outputCol + filter_c;
      image_c = image_c > 0 ? image_c :0;
      image_c = image_c < numCols ? image_c : numCols -1;

      int image1DIdx = numCols * image_r + image_c;
      int filter1DIdx = (filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2;

      outputVal += inputChannel[image1DIdx] * sh_filter[filter1DIdx];
    }
  }

  outputChannel[image1DIndexOutput] = outputVal;

}

__device__
int clamp(int a, int lo, int hi){
   int ret;
   ret = lo <= a ? a : lo;
   ret = ret < hi ? ret : hi -1;
   return ret;
}

__global__
void gaussian_blur_sh_block_image(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO
  
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
  int outputRow = threadIdx.y + blockIdx.y * blockDim.y;
  int outputCol = threadIdx.x + blockIdx.x * blockDim.x;

  unsigned int image1DIndexOutput = outputRow * numCols + outputCol;

  extern __shared__ unsigned char sh_img_block[];

  //shared memory is a block of square of size = blocksize + filterWidth -1;
  int sh_mem_block_size = blockDim.x + filterWidth-1;
  int halfWidth = filterWidth/2;

  int ori_image_row_pos = outputRow - halfWidth;
  int ori_image_col_pos = outputCol - halfWidth;

  int sh_img_row_pos = threadIdx.y;
  int sh_img_col_pos = threadIdx.x;

  ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
  ori_image_col_pos = clamp(ori_image_col_pos, 0, numCols);

  // current pixel mapped to shared_memory location!
  sh_img_block[sh_img_row_pos * sh_mem_block_size + sh_img_col_pos] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];

  int boundary_thresh_row = blockDim.y - filterWidth+1;
  int boundary_thresh_col = blockDim.x - filterWidth+1;

  if (threadIdx.y >= boundary_thresh_row){
    // filling in the lower bottom rectangle of size (filterWidth -1 ) * (blockDim.x) of shared memory except for bottom-right corner
    ori_image_row_pos = outputRow + halfWidth;
    ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
    sh_img_block[(sh_img_row_pos + filterWidth -1)* sh_mem_block_size + sh_img_col_pos] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];
  }

  if (threadIdx.x >= boundary_thresh_col){
    // filling in the right left rectangle of size (blockDim.y) * (filterWidth - 1) of shared memory except for bottom-right corner
    ori_image_row_pos = outputRow - halfWidth;
    ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
    ori_image_col_pos = outputCol + halfWidth;
    ori_image_col_pos = clamp(ori_image_col_pos, 0, numCols);
    sh_img_block[sh_img_row_pos * sh_mem_block_size + sh_img_col_pos + filterWidth-1] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];
  }
  
  if (threadIdx.x < (filterWidth - 1) && threadIdx.y < (filterWidth-1)) {
    // filling the bottom-right corner of zie (filterWidth - 1) ^ (2).
    ori_image_row_pos = outputRow - halfWidth + blockDim.y;
    ori_image_col_pos = outputCol - halfWidth + blockDim.x;

    ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
    ori_image_col_pos = clamp(ori_image_col_pos, 0, numCols);

    sh_img_block[(sh_img_row_pos + blockDim.y)*sh_mem_block_size + sh_img_col_pos + blockDim.x] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];
  }

  __syncthreads();

  if (outputRow >= numRows || outputCol >=numCols){
    return;
  }
  
  float outputVal = 0.0f;

  for (int filter_r = 0; filter_r < filterWidth; filter_r++) {
    for (int filter_c = 0; filter_c < filterWidth; filter_c++) {
      int image_r, image_c;
      image_r = threadIdx.y + filter_r;
      image_c = threadIdx.x + filter_c;
      float imageVal = static_cast<float>(sh_img_block[image_r * sh_mem_block_size + image_c]);
      //float imageVal = sh_img_block[image_r * sh_mem_block_size + image_c]; 
      outputVal += filter[filter_r * filterWidth + filter_c] * imageVal;
      //outputVal += sh_img_block[image_r * sh_mem_block_size + image_c] * filter[filter_r * filterWidth + filter_c];
    }
  }

  outputChannel[image1DIndexOutput] = outputVal;

}


// Function with both filter and image block moved to shared memory!!
__global__
void gaussian_blur_sh_filter_image(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO
  
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
  int outputRow = threadIdx.y + blockIdx.y * blockDim.y;
  int outputCol = threadIdx.x + blockIdx.x * blockDim.x;

  unsigned int image1DIndexOutput = outputRow * numCols + outputCol;

  int sh_mem_block_size = blockDim.x + filterWidth-1;

  extern __shared__ int sh_mem[];
  int *sh_pointer = sh_mem;

  float *sh_filter = (float*)&sh_pointer[0]; // first portion of memory starts from position to 0 and given to filter
  unsigned char *sh_img_block = (unsigned char*)&sh_filter[filterWidth*filterWidth]; // fW * fW * 4 bytes positions taken by filter

  //shared memory is a block of square of size = blocksize + filterWidth -1;
  int halfWidth = filterWidth/2;

  int ori_image_row_pos = outputRow - halfWidth;
  int ori_image_col_pos = outputCol - halfWidth;

  int sh_img_row_pos = threadIdx.y;
  int sh_img_col_pos = threadIdx.x;

  ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
  ori_image_col_pos = clamp(ori_image_col_pos, 0, numCols);

  // current pixel mapped to shared_memory location!
  sh_img_block[sh_img_row_pos * sh_mem_block_size + sh_img_col_pos] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];

  int boundary_thresh_row = blockDim.y - filterWidth+1;
  int boundary_thresh_col = blockDim.x - filterWidth+1;

  if (threadIdx.y >= boundary_thresh_row){
    // filling in the lower bottom rectangle of size (filterWidth -1 ) * (blockDim.x) of shared memory except for bottom-right corner
    ori_image_row_pos = outputRow + halfWidth;
    ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
    sh_img_block[(sh_img_row_pos + filterWidth -1)* sh_mem_block_size + sh_img_col_pos] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];
  }

  if (threadIdx.x >= boundary_thresh_col){
    // filling in the right left rectangle of size (blockDim.y) * (filterWidth - 1) of shared memory except for bottom-right corner
    ori_image_row_pos = outputRow - halfWidth;
    ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
    ori_image_col_pos = outputCol + halfWidth;
    ori_image_col_pos = clamp(ori_image_col_pos, 0, numCols);
    sh_img_block[sh_img_row_pos * sh_mem_block_size + sh_img_col_pos + filterWidth-1] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];
  }
  
  if (threadIdx.x < (filterWidth - 1) && threadIdx.y < (filterWidth-1)) {
    // filling the bottom-right corner of zie (filterWidth - 1) ^ (2).
    ori_image_row_pos = outputRow - halfWidth + blockDim.y;
    ori_image_col_pos = outputCol - halfWidth + blockDim.x;

    ori_image_row_pos = clamp(ori_image_row_pos, 0, numRows);
    ori_image_col_pos = clamp(ori_image_col_pos, 0, numCols);

    sh_img_block[(sh_img_row_pos + blockDim.y)*sh_mem_block_size + sh_img_col_pos + blockDim.x] = inputChannel[ori_image_row_pos * numCols + ori_image_col_pos];
  }

  if (threadIdx.x < filterWidth && threadIdx.y < filterWidth) {
    int filter1Dpos = threadIdx.y * filterWidth + threadIdx.x;
    sh_filter[filter1Dpos] = filter[filter1Dpos];
  }

  __syncthreads();

  if (outputRow >= numRows || outputCol >=numCols){
    return;
  }
  
  float outputVal = 0.0f;

  for (int filter_r = 0; filter_r < filterWidth; filter_r++) {
    for (int filter_c = 0; filter_c < filterWidth; filter_c++) {
      int image_r, image_c;
      image_r = threadIdx.y + filter_r;
      image_c = threadIdx.x + filter_c;
      float imageVal = static_cast<float>(sh_img_block[image_r * sh_mem_block_size + image_c]);
      //float imageVal = sh_img_block[image_r * sh_mem_block_size + image_c]; 
      outputVal += sh_filter[filter_r * filterWidth + filter_c] * imageVal;
      //outputVal += sh_img_block[image_r * sh_mem_block_size + image_c] * filter[filter_r * filterWidth + filter_c];
    }
  }

  outputChannel[image1DIndexOutput] = outputVal;

}


//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO
  //
  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;

  if (row >= numRows || col >=numCols){
    return;
  }
  
  unsigned int image1DIndex = numCols * row + col;
  uchar4 rgba = inputImageRGBA[image1DIndex];

  redChannel[image1DIndex] = rgba.x;
  greenChannel[image1DIndex] = rgba.y;
  blueChannel[image1DIndex] = rgba.z;

}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc

  unsigned int FILTER_BYTES_SIZE = sizeof(float) * filterWidth * filterWidth;

  checkCudaErrors(hipMalloc(&d_filter, FILTER_BYTES_SIZE));

  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!

  checkCudaErrors(hipMemcpy(d_filter, h_filter, FILTER_BYTES_SIZE, hipMemcpyHostToDevice));

}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
//void your_gaussian_blur(uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  int blockWidth = 16;
  int blocksX, blocksY;
 
  if (numCols%blockWidth==0){
     blocksX = numCols/blockWidth;
  } else {
     blocksX = (numCols/blockWidth) + 1;
  }

  if (numRows%blockWidth==0){
     blocksY = numRows/blockWidth;
  } else {
     blocksY = (numRows/blockWidth) + 1;
  }

  //TODO: Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(blockWidth, blockWidth,1);

  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  const dim3 gridSize(blocksX, blocksY,1);

  //TODO: Launch a kernel for separating the RGBA image into different color channels

  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
//  gpuErrchk();


  //TODO: Call your convolution kernel here 3 times, once for each color channel.

  int SH_MEM_FILTER_BYTES_SIZE = sizeof(float) * filterWidth * filterWidth;
  int SH_MEM_IMAGE_BYTES_SIZE = sizeof(unsigned char) * (blockWidth + filterWidth - 1) * (blockWidth + filterWidth - 1);

  //gaussian_blur_sh_filter<<<gridSize, blockSize, SH_MEM_FILTER_BYTES_SIZE>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  //gaussian_blur_sh_filter<<<gridSize, blockSize, SH_MEM_FILTER_BYTES_SIZE>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  //gaussian_blur_sh_filter<<<gridSize, blockSize, SH_MEM_FILTER_BYTES_SIZE>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);

  // Call when no shared memory is used
  //gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  //gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  //gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);


  // Call when only Image Block is shared
  //gaussian_blur_sh_block_image<<<gridSize, blockSize, SH_MEM_IMAGE_BYTES_SIZE>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  //gaussian_blur_sh_block_image<<<gridSize, blockSize, SH_MEM_IMAGE_BYTES_SIZE>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  //gaussian_blur_sh_block_image<<<gridSize, blockSize, SH_MEM_IMAGE_BYTES_SIZE>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);

  // Call when Filter and Image Block both are shared.
  gaussian_blur_sh_filter_image<<<gridSize, blockSize, SH_MEM_IMAGE_BYTES_SIZE + SH_MEM_FILTER_BYTES_SIZE>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur_sh_filter_image<<<gridSize, blockSize, SH_MEM_IMAGE_BYTES_SIZE + SH_MEM_FILTER_BYTES_SIZE>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur_sh_filter_image<<<gridSize, blockSize, SH_MEM_IMAGE_BYTES_SIZE + SH_MEM_FILTER_BYTES_SIZE>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
